#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include "computeQ.cc"
//#include "parboil.h"

#define MU_THREADS_PER_BLOCK 8192
#define Q_THREADS_PER_BLOCK 512

/************ Definitions from computeQ.cc ************
#define PI   3.1415926535897932384626433832795029f
#define PIx2 6.2831853071795864769252867665590058f

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#define K_ELEMS_PER_GRID 2048

struct kValues {
  float Kx;
  float Ky;
  float Kz;
  float PhiMag;
};
*/


__global__
void dev_ComputePhiMagGPU(const int numK, const float* phiR, const float* phiI, float* phiMag) {
    int indexK = blockIdx.x * MU_THREADS_PER_BLOCK + threadIdx.x;
    if (indexK+16 < numK) {
        float real = phiR[indexK];
        float imag = phiI[indexK];
        phiMag[indexK] = real*real + imag*imag;
        for(int i = 1; i < 16; i++) {
            real = phiR[indexK+i];
            imag = phiI[indexK+i];
            phiMag[indexK+i] = real*real + imag*imag;
        }
    }
}

void ComputePhiMagGPU(int numK, float* &phiR_d, float* &phiI_d, float* &phiMag_d) {
  dim3 dim_grid((numK-1)/MU_THREADS_PER_BLOCK + 1, 1, 1);
  dim3 dim_block(MU_THREADS_PER_BLOCK, 1, 1);

  dev_ComputePhiMagGPU<<<dim_grid, dim_block>>>(numK, phiR_d, phiI_d, phiMag_d);
}

/* Original
inline
void
ComputeQCPU(int numK, int numX,
            struct kValues *kVals,
            float* x, float* y, float* z,
            float *__restrict__ Qr, float *__restrict__ Qi) {
  float expArg;
  float cosArg;
  float sinArg;

  int indexK, indexX;

  // Loop over the space and frequency domains.
  // Generally, numX > numK.
  // Since loops are not tiled, it's better that the loop with the smaller
  // cache footprint be innermost.
  for (indexX = 0; indexX < numX; indexX++) {

    // Sum the contributions to this point over all frequencies
    float Qracc = 0.0f;
    float Qiacc = 0.0f;
    for (indexK = 0; indexK < numK; indexK++) {
      expArg = PIx2 * (kVals[indexK].Kx * x[indexX] +
                       kVals[indexK].Ky * y[indexX] +
                       kVals[indexK].Kz * z[indexX]);

      cosArg = cosf(expArg);
      sinArg = sinf(expArg);

      float phi = kVals[indexK].PhiMag;
      Qracc += phi * cosArg;
      Qiacc += phi * sinArg;
    }
    Qr[indexX] = Qracc;
    Qi[indexX] = Qiacc;
  }
}
*/
__global__
void dev_ComputeQGPU(const int numK, const int numX,
                    struct kValues* kVals_d,
                    const float* x_d, const float* y_d, const float* z_d,
                    float* Qr_d, float* Qi_d) {
    // Find index of voxel assigned to this thread
    int indexX = blockIdx.x * Q_THREADS_PER_BLOCK + threadIdx.x;
    
    // Local vars
    float loc_x, loc_y, loc_z;
    float Qracc = Qr_d[indexX];
    float Qiacc = Qi_d[indexX];
    float expArg;
    float cosArg;
    float sinArg;
    
    __shared__ struct kValues kVals_tile[Q_THREADS_PER_BLOCK];

    if(indexX < numX) {
        loc_x = x_d[indexX];
        loc_y = y_d[indexX];
        loc_z = z_d[indexX];
    } else {
        loc_x = 0;
        loc_y = 0;
        loc_z = 0;
    }
    
    for(int i = 0; i < (numK-1)/Q_THREADS_PER_BLOCK + 1; i++) { // Loop for each tile
        //Collaborative loading
        if(indexX < numX) {
            kVals_tile[threadIdx.x] = kVals_d[i*Q_THREADS_PER_BLOCK+threadIdx.x];
        } else {
            kVals_tile[threadIdx.x].Kx = 0;
            kVals_tile[threadIdx.x].Ky = 0;
            kVals_tile[threadIdx.x].Kz = 0;
            kVals_tile[threadIdx.x].PhiMag = 0;
        }
        __syncthreads(); // sync to ensure tile properly loaded
    
        // Accumulation
        if(indexX < numX) { // Checking data bounds
            for (int indexK = 0; indexK < Q_THREADS_PER_BLOCK; indexK++) {
                expArg = PIx2 * (kVals_tile[indexK].Kx * loc_x +
                                kVals_tile[indexK].Ky * loc_y +
                                kVals_tile[indexK].Kz * loc_z);

                cosArg = cosf(expArg);
                sinArg = sinf(expArg);

                float phi = kVals_tile[indexK].PhiMag;
                Qracc += phi * cosArg;
                Qiacc += phi * sinArg;    
            }
        }
        __syncthreads(); // sync to ensure calcs finish before next load
    }

    // if (indexX == 3072) {
    //     printf("thread: %d, Qracc: %f, Qiacc: %f", indexX, Qracc, Qiacc);
    // }

    if(indexX < numX) {
        Qr_d[indexX] = Qracc;
        Qi_d[indexX] = Qiacc;
    }
}

void ComputeQGPU(const int numK, const int numX,
            struct kValues* &kVals_d,
            float* &x_d, float* &y_d, float* &z_d,
            float* &Qr_d, float* &Qi_d) {
    dim3 dim_grid((numX-1)/Q_THREADS_PER_BLOCK + 1, 1, 1);
    dim3 dim_block(Q_THREADS_PER_BLOCK, 1, 1);

    dev_ComputeQGPU<<<dim_grid, dim_block>>>(numK, numX, kVals_d, x_d, y_d, z_d, Qr_d, Qi_d);
}



void createDataStructsGPU(int numK, int numX, 
                          float* &x, float* &y, float* &z,
                          float* &phiR, float* &phiI,
                          float* &x_d, float* &y_d, float* &z_d,
                          float* &phiR_d, float* &phiI_d, float* &phiMag_d,
                          float* &Qr_d, float* &Qi_d, struct kValues* &kVals_d) {
    hipError_t cuda_ret;

    cuda_ret = hipMalloc((void**) &x_d, numX*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMalloc((void**) &y_d, numX*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMalloc((void**) &z_d, numX*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMalloc((void**) &phiR_d, numK*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMalloc((void**) &phiI_d, numK*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMalloc((void**) &phiMag_d, numK*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMalloc((void**) &Qi_d, numX*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMalloc((void**) &Qr_d, numX*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMalloc((void**) &kVals_d, numK*sizeof(struct kValues));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    cuda_ret = hipMemcpy(x_d, x, numX*sizeof(float), hipMemcpyHostToDevice);
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMemcpy(y_d, y, numX*sizeof(float), hipMemcpyHostToDevice);
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMemcpy(z_d, z, numX*sizeof(float), hipMemcpyHostToDevice);
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMemcpy(phiR_d, phiR, numK*sizeof(float), hipMemcpyHostToDevice);
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMemcpy(phiI_d, phiI, numK*sizeof(float), hipMemcpyHostToDevice);
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMemset(Qi_d, 0, numX*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    cuda_ret = hipMemset(Qr_d, 0, numX*sizeof(float));
    if (cuda_ret != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();
}
